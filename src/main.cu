#include "hip/hip_runtime.h"
#include <cstdio>
#include <algorithm>
#include <vector>
#include <cassert>
#include <iostream>
#include <fstream>
#include <chrono>
#include <string>

template <typename T>
class TypeArrayWrapper {
public:
    TypeArrayWrapper(const std::vector<T>& data)
    {
        data_ = new T[data.size()];
        size_ = data.size();
        for (size_t i = 0; i != size_; ++i) {
            data_[i] = data[i];
        }
    }

    TypeArrayWrapper(size_t size, T fill)
    {
        size_ = size;
        data_ = new T[size];
        for (size_t i = 0; i != size; ++i) {
            data_[i] = fill;
        }
    }

    bool isOnDevice() const
    {
        return onDevice_;
    }

    T* deviceData() const
    {
        assert(onDevice_);
        return deviceData_;
    }

    void toDevice()
    {
        assert(!onDevice_);
        initDeviceDataIfNeeded();
        hipMemcpy(deviceData_, data_, size_ * sizeof(T), hipMemcpyHostToDevice);
        onDevice_ = true;
    }

    void toHost()
    {
        assert(onDevice_);
        assert(deviceData_);
        hipMemcpy(data_, deviceData_, size_ * sizeof(T), hipMemcpyDeviceToHost);
        onDevice_ = false;
    }

    size_t size() const
    {
        return size_;
    }

    T& operator[](size_t index)
    {
        assert(!onDevice_);
        return data_[index];
    }

    std::vector<T> asVector()
    {
        assert(!onDevice_);
        std::vector<T> result;
        result.resize(size_);
        for (size_t i = 0; i != size_; ++i) {
            result[i] = data_[i];
        }
        return result;
    }

    void clear()
    {
        clearAll();
    }

    ~TypeArrayWrapper()
    {
        clearAll();
    }

private:
    void clearAll()
    {
        if (data_) {
            free(data_);
            data_ = nullptr;
        }
        if (deviceData_) {
            hipFree(deviceData_);
            deviceData_ = nullptr;
        }
    }

    void initDeviceDataIfNeeded()
    {
        if (!deviceData_) {
            hipMalloc(&deviceData_, size_ * sizeof(T));
        }
    }

    T* data_ = nullptr;
    T* deviceData_ = nullptr;
    size_t size_ = 0;

    bool onDevice_ = false;
};

std::vector<int> addPadding(std::vector<int> pixels, size_t height, size_t width, size_t padding)
{
    const auto coordPrev = [&height, &width](const size_t i, const size_t j) -> size_t {
        return i * width + j;
    };
    const auto coordNew = [&height, &width, &padding](const size_t i, const size_t j) -> size_t {
        return i * (width + padding * 2) + j;
    };

    std::vector<int> result;
    result.resize((height + padding * 2) * (width + padding * 2));
    for (size_t i = 0; i != height + padding * 2; ++i) {
        for (size_t j = 0; j != width + padding * 2; ++j) {
            if (padding <= i && padding <= j && i < height + padding && j < width + padding) {
                result[coordNew(i, j)] = pixels[coordPrev(i - padding, j - padding)];
            } else if (i < padding) {
                size_t coordJ = j < padding
                    ? 0
                    : (j >= width + padding ? width - 1 : j - padding);
                result[coordNew(i, j)] = pixels[coordPrev(0, coordJ)];
            } else if (i >= height + padding) {
                size_t coordJ = j < padding
                    ? 0
                    : (j >= width + padding ? width - 1 : j - padding);
                result[coordNew(i, j)] = pixels[coordPrev(height - 1, coordJ)];
            } else {
                result[coordNew(i, j)] = pixels[coordPrev(i - padding, j < padding ? 0 : width - 1)];
            }
        }
    }
    return result;
}

std::vector<int> removePadding(std::vector<int> pixels, size_t height, size_t width, size_t padding)
{
    const auto coordPrev = [&height, &width](const size_t i, const size_t j) -> size_t {
        return i * width + j;
    };
    const auto coordNew = [&height, &width, &padding](const size_t i, const size_t j) -> size_t {
        return i * (width - padding * 2) + j;
    };

    std::vector<int> result;
    result.resize((height - padding * 2) * (width - padding * 2));
    for (size_t i = 0; i != height; ++i) {
        for (size_t j = 0; j != width; ++j) {
            if (padding <= i && padding <= j && i < height - padding && j < width - padding) {
                result[coordNew(i - padding, j - padding)] = pixels[coordPrev(i, j)];
            }
        }
    }
    return result;
}

std::vector<float> createTransformMatrix(size_t size)
{
    assert(size % 2 == 1);
    int p = size / 2;
    float sigma = 1;
    std::vector<float> result;
    result.reserve(size * size);
    for (int x = -p; x <= p; ++x) {
        for (int y = -p; y <= p; ++y) {
            result.push_back(exp(-(x * x + y * y) / (2 * sigma * sigma)));
        }
    }
    float sum = 0;
    for (const auto& elem : result) {
        sum += elem;
    }
    for (auto& elem : result) {
        elem /= sum;
    }
    return result;
}

__global__ void transform(
    const int height,
    const int width,
    const int maskSize,
    const int* const from,
    int* const to,
    const float* const mask)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index / width;
    const int j = index - i * width;
    if (i < height && j < width) {
        const int pad = maskSize / 2;
        if (pad <= i && pad <= j && i < height - pad && j < width - pad) {
            float sum = 0;
            for (int dx = -pad; dx <= pad; ++dx) {
                for (int dy = -pad; dy <= pad; ++dy) {
                    const int maskIndex = (dx + pad) * maskSize + dy + pad;
                    const int fromIndex = (i + dx) * width + (j + dy);
                    sum += mask[maskIndex] * from[fromIndex];
                }
            }
            to[i * width + j] = sum;
        } else {
            to[i * width + j] = from[i * width + j];
        }
    }
}

std::vector<int> applyTransform(std::vector<int> pixels, size_t height, size_t width, size_t maskSize)
{
    const auto matrix = createTransformMatrix(maskSize);

    TypeArrayWrapper matr(matrix);
    matr.toDevice();

    TypeArrayWrapper data(pixels);
    data.toDevice();

    TypeArrayWrapper<int> result(pixels.size(), 0);
    result.toDevice();

    transform<<<(width * height + 511) / 512, 512>>>(height, width, maskSize, data.deviceData(), result.deviceData(), matr.deviceData());

    result.toHost();
    return result.asVector();
}


std::vector<int> applyTrnasformOnCPU(std::vector<int> pixels, size_t height, size_t width, size_t maskSize)
{
    const auto matrix = createTransformMatrix(maskSize);
    std::vector<int> result;
    result.resize(pixels.size(), 0);

    for (size_t index = 0; index != result.size(); ++index) {
        const int i = index / width;
        const int j = index - i * width;
        if (i < height && j < width) {
            const int pad = maskSize / 2;
            if (pad <= i && pad <= j && i < height - pad && j < width - pad) {
                float sum = 0;
                for (int dx = -pad; dx <= pad; ++dx) {
                    for (int dy = -pad; dy <= pad; ++dy) {
                        const int maskIndex = (dx + pad) * maskSize + dy + pad;
                        const int fromIndex = (i + dx) * width + (j + dy);
                        sum += matrix[maskIndex] * pixels[fromIndex];
                    }
                }
                result[i * width + j] = sum;
            } else {
                result[i * width + j] = pixels[i * width + j];
            }
        }
    }

    return result;
}

class Timer {
public:
    Timer(const std::string& tag)
    : tag_(tag)
    , start_(std::chrono::high_resolution_clock::now())
    {}

    ~Timer()
    {
        const auto diff = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_).count();
        std::cerr << "[" << tag_ << "] " << diff << std::endl;
    }

private:
    const std::string tag_;
    const std::chrono::steady_clock::time_point start_;
};

int main(int argc, char** argv)
{
    assert(argc > 3);
    
    const std::string filename = argv[1];
    std::ifstream fin(filename);

    const std::string mode = argv[2];
    assert(mode == "cuda" || mode == "cpu");

    const std::string kernelSize = argv[3];
    const size_t kernel = atoi(kernelSize.c_str());
    assert(kernel % 2 == 1);
    const size_t padding = kernel / 2;

    size_t height, width;
    fin >> height >> width;

    std::cout << height << ' ' << width << '\n';

    for (const auto& channel : {0, 1, 2}) {
        std::vector<int> pixels;
        {
            Timer timer("Reading channel " + std::to_string(channel));
            for (size_t i = 0; i != height; ++i) {
                for (size_t j = 0; j != width; ++j) {
                    int value;
                    fin >> value;
                    pixels.push_back(value);
                }
            }
        }
        {
            Timer timer("Adding padding");
            pixels = addPadding(std::move(pixels), height, width, padding);
        }
        for (int i = 0; i != 10; ++i) {
            Timer timer("Transforming");
            if (mode == "cuda") {
                pixels = applyTransform(
                    std::move(pixels),
                    height + padding * 2,
                    width + padding * 2,
                    padding * 2 + 1);
            } else {
                pixels = applyTrnasformOnCPU(
                    std::move(pixels),
                    height + padding * 2,
                    width + padding * 2,
                    padding * 2 + 1);
            }
        }
        {
            Timer timer("Removing padding");
            pixels = removePadding(std::move(pixels), height + padding * 2, width + padding * 2, padding);
        }

        {
            Timer("Writing channel " + std::to_string(channel));
            for (size_t i = 0; i != height; ++i) {
                for (size_t j = 0; j != width; ++j) {
                    std::cout << pixels[i * width + j] << ' ';
                }
                std::cout << '\n';
            }
        }
    }
}
