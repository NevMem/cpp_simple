#include "hip/hip_runtime.h"
#include <cstdio>
#include <algorithm>
#include <vector>
#include <cassert>

template <typename T>
class TypeArrayWrapper {
public:
    TypeArrayWrapper(const std::vector<T>& data)
    {
        data_ = new T[data.size()];
        size_ = data.size();
        for (size_t i = 0; i != size_; ++i) {
            data_[i] = data[i];
        }
    }

    TypeArrayWrapper(size_t size, T fill)
    {
        size_ = size;
        data_ = new T[size];
        for (size_t i = 0; i != size; ++i) {
            data_[i] = fill;
        }
    }

    bool isOnDevice() const
    {
        return onDevice_;
    }

    T* deviceData() const
    {
        assert(onDevice_);
        return deviceData_;
    }

    void toDevice()
    {
        assert(!onDevice_);
        initDeviceDataIfNeeded();
        hipMemcpy(deviceData_, data_, size_ * sizeof(T), hipMemcpyHostToDevice);
        onDevice_ = true;
    }

    void toHost()
    {
        assert(onDevice_);
        assert(deviceData_);
        hipMemcpy(data_, deviceData_, size_ * sizeof(T), hipMemcpyDeviceToHost);
        onDevice_ = false;
    }

    size_t size() const
    {
        return size_;
    }

    T& operator[](size_t index)
    {
        assert(!onDevice_);
        return data_[index];
    }

    void clear()
    {
        clearAll();
    }

    ~TypeArrayWrapper()
    {
        clearAll();
    }

private:
    void clearAll()
    {
        if (data_) {
            free(data_);
            data_ = nullptr;
        }
        if (deviceData_) {
            hipFree(deviceData_);
            deviceData_ = nullptr;
        }
    }

    void initDeviceDataIfNeeded()
    {
        if (!deviceData_) {
            hipMalloc(&deviceData_, size_ * sizeof(T));
        }
    }

    T* data_ = nullptr;
    T* deviceData_ = nullptr;
    size_t size_ = 0;

    bool onDevice_ = false;
};

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    TypeArrayWrapper<float> x(N, 0), y(N, 0);

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    x.toDevice();
    y.toDevice();

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, x.deviceData(), y.deviceData());

    y.toHost();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = max(maxError, abs(y[i] - 4.0f));
    }
    printf("Max error: %f\n", maxError);
}